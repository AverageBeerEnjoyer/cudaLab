#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include<chrono>
#include <stdio.h>
#include <iostream>
#include <sstream>
#include <fstream>
#include <hiprand/hiprand_kernel.h>
#include "hiprand.h"

using namespace std;

const int matsize = 2000;

__global__
void multiply(float* m1, float* m2, float* res)
{
	for (int i = threadIdx.x * 2; i < (threadIdx.x + 1) * 2; ++i) {
		for (int j = 0; j < matsize; ++j) {
			float sum = 0;
			for (int k = 0; k < matsize; ++k) {
				sum += (m1[i * matsize + k] * m2[k * matsize + j]);
			}
			res[i * matsize + j] = sum;
		}
	}
}

__global__
void randomMatrix(float* m1, float* m2) {
	hiprandState st;
	int id = threadIdx.x;
	hiprand_init(id, id, 0, &st);
	for (int i = threadIdx.x * 2; i < (threadIdx.x + 1) * 2; ++i) {
		for (int j = 0; j < matsize; ++j) {
			float f = hiprand_uniform(&st);
			m1[i * matsize + j] = f;
		}
	}
	for (int i = threadIdx.x * 2; i < (threadIdx.x + 1) * 2; ++i) {
		for (int j = 0; j < matsize; ++j) {
			float f = hiprand_uniform(&st);
			m2[i * matsize + j] = f;
		}
	}
}
void dispose(float* m1, float* m2, float* res) {
	hipFree(m1);
	hipFree(m2);
	hipFree(res);
}
int multiplyWithCuda(float* m1, float* m2, float* res) {
	float* dev_m1;
	float* dev_m2;
	float* dev_res;

	m1[12] = 12;


	hipError_t err;

	if (err = hipMalloc((void**)&dev_m1, sizeof(float) * matsize * matsize)) {
		cout << "malloc m1 error" << endl;
		dispose(dev_m1, dev_m2, dev_res);
		return 1;
	}
	if (err = hipMalloc((void**)&dev_m2, sizeof(float) * matsize * matsize)) {
		cout << "malloc m2 error" << endl;
		dispose(dev_m1, dev_m2, dev_res);
		return 1;
	}
	if (err = hipMalloc((void**)&dev_res, sizeof(float) * matsize * matsize)) {
		cout << "malloc res error" << endl;
		dispose(dev_m1, dev_m2, dev_res);
		return 1;
	}
	randomMatrix << <1, 1000 >> > (dev_m1, dev_m2);
	if (err = hipDeviceSynchronize()) {
		cout << "cant sync" << endl;
		dispose(dev_m1, dev_m2, dev_res);
		return 1;
	}
	multiply<<<1,1000>>>(dev_m1, dev_m2, dev_res);
	if (err = hipMemcpy(m1, dev_m1, matsize * matsize * sizeof(float), hipMemcpyDeviceToHost)) {
		cout << "copy m1 error" << endl;
		dispose(dev_m1, dev_m2, dev_res);
		return 1;
	}
	if (err = hipMemcpy(m2, dev_m2, matsize * matsize * sizeof(float), hipMemcpyDeviceToHost)) {
		cout << "copy m2 error" << endl;
		dispose(dev_m1, dev_m2, dev_res);
		return 1;
	}
	if (err = hipMemcpy(res, dev_res, matsize * matsize * sizeof(float), hipMemcpyDeviceToHost)) {
		cout << "copy res error" << endl;
		dispose(dev_m1, dev_m2, dev_res);
		return 1;
	}
	dispose(dev_m1, dev_m2, dev_res);

	return 0;
}

void report(chrono::milliseconds t) {
	fstream ofs = fstream("output.txt", ios_base::out);
	stringstream ss;
	ss << "matrix size: " << matsize <<" x " <<matsize<< endl;
	ss << "total time: " << (float)t.count() / 1000 << endl;
	ofs << ss.str();
	ofs.flush();
	ofs.close();
}
int main() {
	float* m1 = new float[matsize * matsize];
	float* m2 = new float[matsize * matsize];
	float* res = new float[matsize * matsize];

	hipSetDevice(0);
	auto start = chrono::system_clock::now();
	multiplyWithCuda(m1, m2, res);
	auto dur = chrono::duration_cast<chrono::milliseconds> (chrono::system_clock::now() - start);

	report(dur);
	return 0;
}